#include "hip/hip_runtime.h"
#include "./relu.h"
#include <iostream>
#include <math.h>


static void HandleError(hipError_t err, const char *file, int line);
inline void error_check(hipError_t err, const char* file, int line);
//__global__ void relu(float* mat, float* result);
__global__ void relu(float* mat, int mat_size, int rows, int cols);
void getThrCnt(size_t* thrCnt);
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

float cpu_time2(timespec* start, timespec* end){
	/**
	 * Function responsible for returning the ellapsed time in
	 * milliseconds
	 */
	return ((1e9*end->tv_sec + end->tv_nsec) - (1e9*start->tv_sec + 
	start->tv_nsec))/1e6;
}

// void ReLU::forward(const Matrix& bottom) {

//   int rows = bottom.rows();
//   int cols = bottom.cols();

//   int mat_size = rows * cols;

//   // a = z*(z>0)
//   timespec ts, te;
//   clock_gettime(CLOCK_MONOTONIC_RAW, &ts);
//   top = bottom.cwiseMax(0.0);

//   // End timing the CPU implementation
//   clock_gettime(CLOCK_MONOTONIC_RAW, &te);
//   std::cout << "Matrix Size: " << mat_size << " | elapsed time: " << cpu_time2(&ts, &te) << std::endl;
// }

void ReLU::forward(const Matrix& bottom) {
  // get rows and cols
  int rows = bottom.rows();
  int cols = bottom.cols();

  int mat_size = rows * cols;
  int mat_mem_size = mat_size * sizeof(float);

  float *d_mat;
 
  HANDLE_ERROR(hipMalloc((void **)&d_mat, mat_mem_size));

  CUDA_CHECK(hipGetLastError());

  HANDLE_ERROR(hipMemcpy(d_mat, bottom.data(), mat_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipGetLastError());

  float num_threadsX = 32.00;
  float num_threadsY = 32.00;

  int dimGridSizeX = ceil((float)cols/num_threadsX);
  int dimGridSizeY = ceil((float)rows/num_threadsY);

  // get dimensions
  dim3 DimGrid(dimGridSizeX, dimGridSizeY, 1);
  dim3 DimBlock(num_threadsX, num_threadsY, 1);

  CUDA_CHECK(hipGetLastError());

  // TIMING:
  hipEvent_t start, stop; //declare a start and stop event
  HANDLE_ERROR(hipEventCreate(&start)); //create both events
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start)); //insert the start event into the stream

  relu<<<DimGrid, DimBlock>>>(d_mat, mat_size, rows, cols);

  // End profiling code:
  HANDLE_ERROR(hipEventRecord(stop)); //insert the stop event into the stream
  hipDeviceSynchronize();
  float milliseconds = 0; //declare a variable to store runtime
  HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop)); //get the elapsed

  CUDA_CHECK(hipGetLastError());

  float * result = (float*) malloc(mat_size * sizeof(float));

  HANDLE_ERROR(hipMemcpy(result, d_mat, mat_mem_size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipGetLastError());

  Matrix output = Eigen::Map<Matrix>(result, rows, cols);
  top = output;

  // Free Cuda Memory
  HANDLE_ERROR(hipFree(d_mat));
  CUDA_CHECK(hipGetLastError());
} // END FORWARD

//__global__ void relu(float* mat, float* result){
__global__ void relu(float* mat, int mat_size, int rows, int cols){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = rows * col + row;

  if (idx < mat_size){
    mat[idx] = mat[idx] * (int)(mat[idx] > 0);
  }
}

void ReLU::backward(const Matrix& bottom, const Matrix& grad_top) {
  // d(L)/d(z_i) = d(L)/d(a_i) * d(a_i)/d(z_i)
  //             = d(L)/d(a_i) * 1*(z_i>0)
  Matrix positive = (bottom.array() > 0.0).cast<float>();
  grad_bottom = grad_top.cwiseProduct(positive);
}

// Handle Error

static void HandleError(hipError_t err, const char *file, int line ) {
	/**
	 * Handle error macro provided by instructor for cuda library calls
	 */
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line );
	}
}

inline void error_check(hipError_t err, const char* file, int line) {
    if(err != hipSuccess) {
        ::fprintf(stderr, "CUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(err));
        abort();
    }
}

void getThrCnt(size_t* thrCnt) {

  HANDLE_ERROR(hipFree(0));

  int dev = 0;

  HANDLE_ERROR(hipGetDevice(&dev));

  HANDLE_ERROR(hipSetDevice(dev));

  // Find maximum threads per block dimension and use that
  hipDeviceProp_t prop;

  HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));

  (*thrCnt) = (int)sqrt((double)prop.maxThreadsDim[0]);

}
