#include "hip/hip_runtime.h"
#include "./fully_connected.h"
#include <iostream>

// SK: CUDA error handling functions
static void HandleError(hipError_t err, const char *file, int line);
inline void error_check(hipError_t err, const char* file, int line);
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

// SK: Forward Prop Fully Connected Kernel
#define TILE_SIZE 16
void getDims(size_t* thrCnt, size_t* blkRWs, size_t* blkCLs, size_t rows, size_t cols);
__global__ void d_transpose(float* a, float* b, int rowsA, int colsA);
__global__ void d_flip_str_order(float* row_mtx, float* col_mtx, size_t rows, size_t cols, bool isRowMajor);
__global__ void gpu_mat_mul(float *d_A, float *d_B, float *d_C, int m, int n, int k);

// Return the result of row * width + col
__host__ __device__ size_t indexify_RM(size_t row, size_t width, size_t col) {
  return ((row * width) + col);
}

// Return the result of col * height + row
__host__ __device__ size_t indexify_CM(size_t row, size_t height, size_t col) {
  return ((col * height) + row);
}

void getThrCnt(size_t* thrCnt) {
  HANDLE_ERROR(hipFree(0));
  int dev = 0;
  HANDLE_ERROR(hipGetDevice(&dev));
  HANDLE_ERROR(hipSetDevice(dev));
  // Find maximum threads per block dimension and use that
  hipDeviceProp_t prop;
  HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));
  
  (*thrCnt) = (int)sqrt((double)prop.maxThreadsDim[0]);
}

void FullyConnected::init() {
  weight.resize(dim_in, dim_out);
  bias.resize(dim_out);
  grad_weight.resize(dim_in, dim_out);
  grad_bias.resize(dim_out);
  set_normal_random(weight.data(), weight.size(), 0, 0.01);
  set_normal_random(bias.data(), bias.size(), 0, 0.01);
}

// SK: Original CPU implementation (without timing)
// void FullyConnected::forward(const Matrix& bottom) {
//   // z = w' * x + b
//   const int n_sample = bottom.cols();
//   top.resize(dim_out, n_sample);
//   top = weight.transpose() * bottom;
//   top.colwise() += bias;
// }

// SK: Modified Parallel implementation
void FullyConnected::forward(const Matrix& bottom) {
  // z = w' * x + b
  // Resize output matrix so weight * bottom is possible
  const int n_sample = bottom.cols();
  top.resize(dim_out, n_sample); // output matrix will be ROWS[dim_out] x COLS[n_sample]

  size_t sz_weight = weight.size() * sizeof(float);
  size_t sz_bottom = bottom.size() * sizeof(float);
  size_t sz_top = top.size() * sizeof(float);

  float *d_weight, *d_bottom, *d_top, *h_top, *h_wght;
  h_top = (float*)malloc(sz_top);
  h_wght = (float*)malloc(sz_weight);
  
  HANDLE_ERROR(hipMalloc((void**)&d_weight, sz_weight));
  HANDLE_ERROR(hipMalloc((void**)&d_bottom, sz_bottom));
  HANDLE_ERROR(hipMalloc((void**)&d_top, sz_top));

  HANDLE_ERROR(hipMemcpy(d_weight, weight.data(), sz_weight, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_bottom, bottom.data(), sz_bottom, hipMemcpyHostToDevice));

  size_t threadCnt = 0;
  getThrCnt(&threadCnt);
  float num_threads = (float)threadCnt;

  int dimGridSizeY_C = ceil((float)top.rows()/num_threads);
	int dimGridSizeX_C = ceil((float)top.cols()/num_threads);
  dim3 DimGrid(dimGridSizeX_C, dimGridSizeY_C, 1);
  dim3 DimBlock(num_threads, num_threads, 1);
  
  gpu_mat_mul<<<DimGrid, DimBlock>>>(d_weight, d_bottom, d_top, dim_out, dim_in, n_sample);

  HANDLE_ERROR(hipMemcpy(h_top, d_top, sz_top, hipMemcpyDeviceToHost));

  top = Eigen::Map<Vector>(h_top, dim_out * n_sample);
  top.resize(dim_out, n_sample);

  top.colwise() += bias;  // for each column in top, add vector bias

  free(h_top);
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_bottom));
  HANDLE_ERROR(hipFree(d_top));
  CUDA_CHECK(hipGetLastError());
}

// =======================================================
//                   Device Operations
// =======================================================

// Transpose Matrix A to matrix B
__global__ void d_transpose(float* a, float* b, int rowsA, int colsA) {
   int rIDX = threadIdx.x + blockIdx.x * blockDim.x;
   int cIDX = threadIdx.y + blockIdx.y * blockDim.y;

  // Transpose
  // ensure tIdxs are within the weight matrix dimensions
  if ( rIDX < rowsA && cIDX < colsA) {
    int m_idx = cIDX * rowsA + rIDX;
    int t_idx = rIDX * colsA + cIDX;
    b[t_idx] = a[m_idx];
  }
}

__global__ void gpu_mat_mul(float *d_A, float *d_B, float *d_C, int m, int n, int k){
	/**
	 * Function responsible for performing GPU matrix multiplication w/o shared memory on d_A and d_B
	 * and storing the result in d_C.
	 */
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int a_idx, b_idx, c_idx;
	if(row < m && col < k){
		int i;
		for(i = 0; i < n; i++){
      // row * width + col
      // col * height + row
      //a_idx = i * m + row;
			b_idx = col * n + i;
			c_idx = col * m + row;
			a_idx = row * n + i;
			// b_idx = i * k + col;
			// c_idx = row * k + col;
			d_C[c_idx] += d_A[a_idx] * d_B[b_idx];
		}
	}
}

// =======================================================
//                 End Device Operations
// =======================================================

void FullyConnected::backward(const Matrix& bottom, const Matrix& grad_top) {
  const int n_sample = bottom.cols();
  // d(L)/d(w') = d(L)/d(z) * x'
  // d(L)/d(b) = \sum{ d(L)/d(z_i) }
  grad_weight = bottom * grad_top.transpose();
  grad_bias = grad_top.rowwise().sum();
  // d(L)/d(x) = w * d(L)/d(z)
  grad_bottom.resize(dim_in, n_sample);
  grad_bottom = weight * grad_top;
}

void FullyConnected::update(Optimizer& opt) {
  Vector::AlignedMapType weight_vec(weight.data(), weight.size());
  Vector::AlignedMapType bias_vec(bias.data(), bias.size());
  Vector::ConstAlignedMapType grad_weight_vec(grad_weight.data(),
                                              grad_weight.size());
  Vector::ConstAlignedMapType grad_bias_vec(grad_bias.data(), grad_bias.size());

  opt.update(weight_vec, grad_weight_vec);
  opt.update(bias_vec, grad_bias_vec);
}

std::vector<float> FullyConnected::get_parameters() const {
  std::vector<float> res(weight.size() + bias.size());
  // Copy the data of weights and bias to a long vector
  std::copy(weight.data(), weight.data() + weight.size(), res.begin());
  std::copy(bias.data(), bias.data() + bias.size(),
            res.begin() + weight.size());
  return res;
}

void FullyConnected::set_parameters(const std::vector<float>& param) {
  if (static_cast<int>(param.size()) != weight.size() + bias.size())
      throw std::invalid_argument("Parameter size does not match");
  std::copy(param.begin(), param.begin() + weight.size(), weight.data());
  std::copy(param.begin() + weight.size(), param.end(), bias.data());
}

std::vector<float> FullyConnected::get_derivatives() const {
  std::vector<float> res(grad_weight.size() + grad_bias.size());
  // Copy the data of weights and bias to a long vector
  std::copy(grad_weight.data(), grad_weight.data() + grad_weight.size(),
            res.begin());
  std::copy(grad_bias.data(), grad_bias.data() + grad_bias.size(),
            res.begin() + grad_weight.size());
  return res;
}

static void HandleError(hipError_t err, const char *file, int line ) {
	/**
	 * Handle error macro provided by instructor for cuda library calls
	 */
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line );
	}
}

inline void error_check(hipError_t err, const char* file, int line) {
    if(err != hipSuccess) {
        ::fprintf(stderr, "CUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(err));
        abort();
    }
}

void getDims(size_t* thrCnt, size_t* blkRWs, size_t* blkCLs, size_t rows, size_t cols) {
   HANDLE_ERROR(hipFree(0));
   int dev = 0;
   HANDLE_ERROR(hipGetDevice(&dev));
   HANDLE_ERROR(hipSetDevice(dev));
   // Find maximum threads per block dimension and use that
   hipDeviceProp_t prop;
   HANDLE_ERROR(hipGetDeviceProperties(&prop, dev));
   
   (*thrCnt) = (int)sqrt((double)prop.maxThreadsDim[0]);
   (*blkRWs) = ((rows+(*thrCnt)-1) / (*thrCnt));
   (*blkCLs) = ((cols+(*thrCnt)-1) / (*thrCnt));
}